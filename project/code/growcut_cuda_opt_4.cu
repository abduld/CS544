#include "hip/hip_runtime.h"

#include <limits.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <fstream>

#include "assert.h"
#include "dataset.h"
#include "mfi.h"
#include "timer.h"

#include "math.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define RADIUS 1
#define MAX_ITERATIONS 2048 / 32
#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8

template <typename T0, typename T1> static T0 zCeil(const T0 &n, const T1 &d) {
  return (T0) ceil(static_cast<double>(n) / static_cast<double>(d));
}

#define cudaCheck(stmt) _cudaCheck(__LINE__, stmt)
#define _cudaCheck(line, stmt)                                                 \
  do {                                                                         \
    hipError_t p_err = stmt;                                                  \
    assert(p_err == hipSuccess);                                              \
    if (p_err != hipSuccess) {                                                \
      printf("ERROR on line %d (%s): %s -- %d\n", line, #stmt,                 \
             hipGetErrorString(p_err), p_err);                                \
    }                                                                          \
  } while (0)

#define _max(x, y) (((x) > (y)) ? x : y)
#define _min(x, y) (((x) < (y)) ? x : y)
#define g(x, y) (1 - _max(x, y) - _min(x, y) / 2)

__global__ void growcut(unsigned short const *__restrict__ img, char *label,
                        float *strength, int height, int width) {

  __shared__ unsigned short
      imgShared[BLOCKDIM_Y + 2 * RADIUS][BLOCKDIM_X + 2 * RADIUS];
  __shared__ char lShared[BLOCKDIM_Y + 2 * RADIUS][BLOCKDIM_X + 2 * RADIUS];
  __shared__ float sShared[BLOCKDIM_Y + 2 * RADIUS][BLOCKDIM_X + 2 * RADIUS];
  __shared__ char ltShared[BLOCKDIM_Y][BLOCKDIM_X];
  __shared__ float stShared[BLOCKDIM_Y][BLOCKDIM_X];

  int tidX = threadIdx.x;
  int tidY = threadIdx.y;
  int tx = tidX;
  int ty = tidY;
  int jj = tidX + BLOCKDIM_X * blockIdx.x;
  int ii = tidY + BLOCKDIM_Y * blockIdx.y;

#define idx(arry, y, x)                                                        \
  ((y) >= 0 && (y) < height && (x) >= 0 && (x) < width)                        \
      ? arry[(y) * width + (x)]                                                \
      : 0

  tidY += RADIUS;
  tidX += RADIUS;

  imgShared[tidY][tidX] = idx(img, ii, jj);
  sShared[tidY][tidX] = idx(strength, ii, jj);
  lShared[tidY][tidX] = idx(label, ii, jj);

  if (tx < RADIUS) {
    imgShared[tidY][tidX - RADIUS] = idx(img, ii, jj - RADIUS);
    sShared[tidY][tidX - RADIUS] = idx(strength, ii, jj - RADIUS);
    lShared[tidY][tidX - RADIUS] = idx(label, ii, jj - RADIUS);

    imgShared[tidY][tidX + BLOCKDIM_X] = idx(img, ii, jj + BLOCKDIM_X);
    sShared[tidY][tidX + BLOCKDIM_X] = idx(strength, ii, jj + BLOCKDIM_X);
    lShared[tidY][tidX + BLOCKDIM_X] = idx(label, ii, jj + BLOCKDIM_X);
  }

  if (ty < RADIUS) {
    imgShared[tidY - RADIUS][tidX] = idx(img, ii - RADIUS, jj);
    sShared[tidY - RADIUS][tidX] = idx(strength, ii - RADIUS, jj);
    lShared[tidY - RADIUS][tidX] = idx(label, ii - RADIUS, jj);

    imgShared[tidY + BLOCKDIM_Y][tidX] = idx(img, ii + BLOCKDIM_Y, jj);
    sShared[tidY + BLOCKDIM_Y][tidX] = idx(strength, ii + BLOCKDIM_Y, jj);
    lShared[tidY + BLOCKDIM_Y][tidX] = idx(label, ii + BLOCKDIM_Y, jj);
  }

  for (int kk = 0; kk < 16; kk++) {
    __syncthreads();
    char nl = 0;
    float ns = 0;
    if (jj < width && ii < height) {
      char lq;
      float sq, gc;
      unsigned short cq;

      unsigned short cp = imgShared[tidY][tidX];
      char lp = lShared[tidY][tidX];
      float sp = sShared[tidY][tidX];
      nl = lp;
      ns = sp;

      cq = imgShared[tidY - 1][tidX];
      lq = lShared[tidY - 1][tidX];
      sq = sShared[tidY - 1][tidX];
      gc = g(cp, cq) * sq;
      if (gc > sp) {
        nl = lq;
        ns = gc;
      }

      cq = imgShared[tidY + 1][tidX];
      lq = lShared[tidY + 1][tidX];
      sq = sShared[tidY + 1][tidX];
      gc = g(cp, cq) * sq;
      if (gc > sp) {
        nl = lq;
        ns = gc;
      }

      cq = imgShared[tidY][tidX - 1];
      lq = lShared[tidY][tidX - 1];
      sq = sShared[tidY][tidX - 1];
      gc = g(cp, cq) * sq;
      if (gc > sp) {
        nl = lq;
        ns = gc;
      }

      cq = imgShared[tidY][tidX + 1];
      lq = lShared[tidY][tidX + 1];
      sq = sShared[tidY][tidX + 1];
      gc = g(cp, cq) * sq;
      if (gc > sp) {
        nl = lq;
        ns = gc;
      }

    }

    ltShared[ty][tx] = nl;
    stShared[ty][tx] = ns;

    __syncthreads();

    lShared[tidY][tidX] = ltShared[ty][tx];
    sShared[tidY][tidX] = stShared[ty][tx];

    if (tx < RADIUS) {
      sShared[tidY][tidX - RADIUS] =
          (sShared[tidY][tidX - RADIUS] + stShared[ty][tx]) / 2;
      lShared[tidY][tidX - RADIUS] =
          (lShared[tidY][tidX - RADIUS] + ltShared[ty][tx]) / 2;

      sShared[tidY][tidX + BLOCKDIM_X] =
          (sShared[tidY][tidX + BLOCKDIM_X] +
           stShared[ty][BLOCKDIM_X - tx - RADIUS]) / 2;
      lShared[tidY][tidX + BLOCKDIM_X] =
          (lShared[tidY][tidX + BLOCKDIM_X] +
           ltShared[ty][BLOCKDIM_X - tx - RADIUS]) / 2;
    }

    if (ty < RADIUS) {
      sShared[tidY - RADIUS][tidX] =
          (sShared[tidY - RADIUS][tidX] + stShared[ty][tx]) / 2;
      lShared[tidY - RADIUS][tidX] =
          (lShared[tidY - RADIUS][tidX] + ltShared[ty][tx]) / 2;

      sShared[tidY + BLOCKDIM_Y][tidX] =
          (sShared[tidY + BLOCKDIM_Y][tidX] +
           stShared[BLOCKDIM_Y - ty - RADIUS][tx]) / 2;
      lShared[tidY + BLOCKDIM_Y][tidX] =
          (lShared[tidY + BLOCKDIM_Y][tidX] +
           ltShared[BLOCKDIM_Y - ty - RADIUS][tx]) / 2;
    }

  }
  if (jj < width && ii < height) {
    label[ii * width + jj] = ltShared[ty][tx];
    strength[ii * width + jj] = stShared[ty][tx];
  }
  return;
}

int runGrowCut(MFI *mfi, char *label, int *iterations0) {
  int iterations = 0;
  int width = mfi->width;
  int height = mfi->height;
  char *nextLabel = (char *)malloc(sizeof(char) * width * height);
  float *strength = (float *)malloc(sizeof(float) * width * height);
  float *nextStrength = (float *)malloc(sizeof(float) * width * height);
  unsigned short *cap_source = (unsigned short *)mfi->cap_source;
  unsigned short *cap_sink = (unsigned short *)mfi->cap_sink;
  int len = width * height;
  for (int ii = 0; ii < len; ii++) {
    float s = label[ii] != 0;
    strength[ii] = s;
  }

  unsigned short *dcapsource;
  float *dStrength;
  char *dLabel;

  cudaCheck(hipMalloc(&dcapsource, sizeof(unsigned short) * len));
  cudaCheck(hipMalloc(&dStrength, sizeof(float) * len));
  cudaCheck(hipMalloc(&dLabel, sizeof(char) * len));

  cudaCheck(hipMemcpy(dcapsource, mfi->cap_source,
                       sizeof(unsigned short) * len, hipMemcpyHostToDevice));
  cudaCheck(
      hipMemcpy(dLabel, label, sizeof(char) * len, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dStrength, strength, sizeof(float) * len,
                       hipMemcpyHostToDevice));

  dim3 blockDim(BLOCKDIM_X, BLOCKDIM_Y);
  dim3 gridDim(zCeil(width, blockDim.x), zCeil(height, blockDim.x));

  while (iterations++ < MAX_ITERATIONS) {
    growcut << <gridDim, blockDim>>>
        (dcapsource, dLabel, dStrength, height, width);
    cudaCheck(hipDeviceSynchronize());
  }

  cudaCheck(
      hipMemcpy(label, dLabel, sizeof(char) * len, hipMemcpyDeviceToHost));

  hipFree(dcapsource);
  hipFree(dStrength);
  hipFree(dLabel);

  free(nextLabel);
  free(strength);
  free(nextStrength);

  *iterations0 = iterations;
  return -1;
}

int main(int argc, char **argv) {

  const char *dataset_path =
      argc == 2 ? argv[1] : "C:\\Users\\abduld\\Documents\\visual studio "
                            "2012\\Projects\\growcut\\x64\\Debug\\dataset";

  int num_instances = (sizeof(instances) / sizeof(Instance));

  ofstream timesFile;
  string timeFileName = string(dataset_path);
  timeFileName.append("\\..\\times_cuda_opt_c2070_alpha_max_itermore.data");
  timesFile.open(timeFileName, ios::out);
  timesFile << "instance,num,width,height,changes,iterations,time\n";

  for (int i = 0; i < num_instances; i++) {

    for (int j = 0; j < instances[i].count; j++) {
      char fileName[1024];

      ofstream myfile;
      string sfileName;
      int iterations;
      unsigned short *cap_source;
      unsigned short *cap_sink;
      char *label;
      int changes;
      uint64_t tic, toc;
      double compute_time;

      sprintf(fileName, instances[i].filename, dataset_path, j);
      MFI *mfi = mfi_read(fileName);

      if (!mfi) {
        //printf("FAILED to read instance %s\n",fileName);
        goto skip;
      }

      if (mfi->connectivity != 4 || mfi->dimension != 2 ||
          mfi->type_terminal_cap != MFI::TYPE_UINT16 ||
          mfi->type_neighbor_cap != MFI::TYPE_UINT16) {
        goto skip;
      }

      cap_source = (unsigned short *)mfi->cap_source;
      cap_sink = (unsigned short *)mfi->cap_sink;
      label = (char *)calloc(mfi->width * mfi->height, sizeof(char));
      for (int ii = 0; ii < mfi->width * mfi->height; ii++) {
        if (cap_source[ii] > 15000) {
          label[ii] = 1;
        }
        if (cap_sink[ii] > 15000) {
          label[ii] = -1;
        }
      }

      tic = _hrtime();
      changes = runGrowCut(mfi, label, &iterations);
      toc = _hrtime();

      compute_time = (toc - tic) / 1000000000.0f;

      sfileName = string(fileName);
      sfileName.append(".dat");

      myfile.open(sfileName, ios::out);

      for (int ii = 0; ii < mfi->height; ii++) {
        for (int jj = 0; jj < mfi->width; jj++) {
          myfile << ((int) label[ii * mfi->width + jj] + 2) / 2 << " ";
          //myfile << ((int) cap_sink[ii*mfi->width + jj]) << " ";
        }
        myfile << "\n";
      }
      myfile.close();

      printf("%s,%d,%d,%d,%d,%d,%0.9f \n", instances[i].name, j, mfi->width,
             mfi->height, changes, iterations, compute_time);
      timesFile << instances[i].name << "," << j << "," << mfi->width << ","
                << mfi->height << "," << changes << "," << iterations << ","
                << compute_time << "\n";
      timesFile.flush();
      free(label);
    skip:
      if (mfi != NULL)
        mfi_free(mfi);

    }
  }
  timesFile.close();
  return 0;
}